#include "hip/hip_runtime.h"
// Name: Deepashika Maduwanthi
// Student Id: 1432291
//--------------------------------------------------


/*hese are header files, include <stdio.h> 
-the compiler to include this header file for compilation
stdlib.h header provides variable types,several macros, 
and functions to performe general functions.*/

#include <stdio.h>
#inlude <stdlib.h>

/modify the CUDA_task2 program to generate A and B matrix automatically/

#define N 4

 
/*Global function is also called "kernels".
 It's the functions that you may call from the host side.
 
*/

__global__ void Matri_Add(int A[][N], int B[][N], int C[][N]){
	
	
	 // Thread row and column 
        int i = threadIdx.x;
		int j = threadIdx.y;

		C[i][j] = A [i][j] + B[i][j];

}
 //function type was changed and added new parameter to the function
void randmatfunc(int newmat[N][N]){
  int i, j, k; 
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
          k = rand() % 100 + 1;;
            printf("%d ", k);
            newmat[i][j] =k;
        }
        printf("\n");
    }
}
/*To generate A and B matrix automatically, code was changes as below(Remove matrix numbers)  */

int main(){

int A[N][N];  
randmatfunc(A);
  
int B[N][N];  
randmatfunc(B);  

int C[N][N];

//calling the poniters
  int (*d_A)[N], (*d_B)[N], (*d_C)[N];

// allocate device copies of A,B, C	
  hipMalloc((void**)&d_A, (N*N)*sizeof(int));
  hipMalloc((void**)&d_B, (N*N)*sizeof(int));
  hipMalloc((void**)&d_C, (N*N)*sizeof(int));

 // CUDA memory copy types(copy input to device from host)
  hipMemcpy(d_A, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, (N*N)*sizeof(int), hipMemcpyHostToDevice);

  int numBlocks = 1;
  
  // N threads (kernel invoke N threads)
  dim3 threadsPerBlock(N,N);
  Matri_Add<<<numBlocks,threadsPerBlock>>>(d_A,d_B,d_C);

// copy result of device back to host 
  hipMemcpy(C, d_C, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

	int i, j; printf("C = \n"); 
	for(i=0;i<N;i++){
	for(j=0;j<N;j++){ 
	printf("%d ", C[i][j]);
	}
	printf("\n");
	}

//  cleanup 
  hipFree(d_A); 
  hipFree(d_B); 
  hipFree(d_C);

  printf("\n");

  return 0;
}
